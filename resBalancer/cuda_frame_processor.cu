#include "hip/hip_runtime.h"
#include "cuda_frame_processor.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <memory>
#include <algorithm>

// CUDA kernel for high-performance bilinear resize
__global__ void cuda_resize_bilinear_kernel(
    const unsigned char* __restrict__ input,
    unsigned char* __restrict__ output,
    int input_width, int input_height,
    int output_width, int output_height,
    int channels, size_t input_pitch, size_t output_pitch) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= output_width || y >= output_height) return;
    
    // Calculate the corresponding position in the input image
    float src_x = (float)x * input_width / output_width;
    float src_y = (float)y * input_height / output_height;
    
    int x1 = (int)src_x;
    int y1 = (int)src_y;
    int x2 = min(x1 + 1, input_width - 1);
    int y2 = min(y1 + 1, input_height - 1);
    
    float fx = src_x - x1;
    float fy = src_y - y1;
    
    // Bilinear interpolation for each channel
    for (int c = 0; c < channels; c++) {
        float val = (1.0f - fx) * (1.0f - fy) * input[(y1 * input_pitch) + x1 * channels + c] +
                    fx * (1.0f - fy) * input[(y1 * input_pitch) + x2 * channels + c] +
                    (1.0f - fx) * fy * input[(y2 * input_pitch) + x1 * channels + c] +
                    fx * fy * input[(y2 * input_pitch) + x2 * channels + c];
        
        output[(y * output_pitch) + x * channels + c] = (unsigned char)__float2int_rn(val);
    }
}

// CUDA kernel for horizontal flip (mirror)
__global__ void cuda_mirror_horizontal_kernel(
    const unsigned char* __restrict__ input,
    unsigned char* __restrict__ output,
    int width, int height, int channels,
    size_t input_pitch, size_t output_pitch) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int flipped_x = width - 1 - x;
    
    // Copy pixels with horizontal flip
    for (int c = 0; c < channels; c++) {
        output[(y * output_pitch) + flipped_x * channels + c] = 
            input[(y * input_pitch) + x * channels + c];
    }
}

// CUDA kernel for Gaussian blur (performance optimization)
__global__ void cuda_gaussian_blur_kernel(
    const unsigned char* __restrict__ input,
    unsigned char* __restrict__ output,
    int width, int height, int channels,
    size_t pitch, float sigma) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // 5x5 Gaussian kernel for sigma ~= 1.0
    const float kernel[5][5] = {
        {0.003765f, 0.015019f, 0.023792f, 0.015019f, 0.003765f},
        {0.015019f, 0.059912f, 0.094907f, 0.059912f, 0.015019f},
        {0.023792f, 0.094907f, 0.150342f, 0.094907f, 0.023792f},
        {0.015019f, 0.059912f, 0.094907f, 0.059912f, 0.015019f},
        {0.003765f, 0.015019f, 0.023792f, 0.015019f, 0.003765f}
    };
    
    for (int c = 0; c < channels; c++) {
        float sum = 0.0f;
        
        for (int ky = -2; ky <= 2; ky++) {
            for (int kx = -2; kx <= 2; kx++) {
                int px = min(max(x + kx, 0), width - 1);
                int py = min(max(y + ky, 0), height - 1);
                
                sum += kernel[ky + 2][kx + 2] * input[(py * pitch) + px * channels + c];
            }
        }
        
        output[(y * pitch) + x * channels + c] = (unsigned char)__float2int_rn(sum);
    }
}

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << std::endl; \
            return false; \
        } \
    } while(0)

#define CUDA_CHECK_VOID(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << std::endl; \
            return; \
        } \
    } while(0)

// CudaFrameProcessor Implementation
CudaFrameProcessor::CudaFrameProcessor(const CudaConfig& cfg) 
    : config(cfg), cuda_context(nullptr), memory_pool(nullptr), 
      memory_pool_size(0), processing_times(nullptr), time_index(0), initialized(false) {
    
    // Initialize CUDA streams array
    for (int i = 0; i < 8; i++) {
        cuda_streams[i] = nullptr;
    }
    
    // Reserve space for frame buffers
    frame_buffers.reserve(10);  // Reserve space for efficiency
}

CudaFrameProcessor::~CudaFrameProcessor() {
    shutdown();
}

bool CudaFrameProcessor::initialize() {
    // Check CUDA device availability
    int device_count = 0;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    
    if (device_count == 0) {
        std::cerr << "No CUDA devices found" << std::endl;
        return false;
    }
    
    // Set device
    if (config.device_id >= device_count) {
        std::cerr << "Invalid device ID: " << config.device_id << std::endl;
        return false;
    }
    
    CUDA_CHECK(hipSetDevice(config.device_id));
    
    // Get device properties for safety checks
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, config.device_id));
    
    std::cout << "Initializing CUDA on device: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Global memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
    
    // Validate memory requirements
    size_t required_memory = config.memory_pool_size_mb * 1024 * 1024;
    if (required_memory > prop.totalGlobalMem * 0.8) { // Use max 80% of GPU memory
        std::cerr << "Requested memory pool too large for device" << std::endl;
        return false;
    }
    
    // Create CUDA streams for concurrent processing
    if (!create_cuda_streams()) {
        std::cerr << "Failed to create CUDA streams" << std::endl;
        return false;
    }
    
    // Allocate memory pools
    if (!allocate_memory_pools()) {
        std::cerr << "Failed to allocate memory pools" << std::endl;
        destroy_cuda_streams();
        return false;
    }
    
    // Allocate performance tracking array
    processing_times = new float[100];
    memset(processing_times, 0, sizeof(float) * 100);
    
    // Enable memory pinning if requested
    if (config.enable_memory_pinning) {
        // This would be implemented for zero-copy transfers
        std::cout << "Memory pinning enabled for zero-copy transfers" << std::endl;
    }
    
    initialized = true;
    std::cout << "✅ CUDA frame processor initialized successfully" << std::endl;
    return true;
}

void CudaFrameProcessor::shutdown() {
    if (!initialized) return;
    
    // Synchronize all streams before cleanup
    for (int i = 0; i < config.max_concurrent_streams; i++) {
        if (cuda_streams[i]) {
            hipStreamSynchronize((hipStream_t)cuda_streams[i]);
        }
    }
    
    // Clean up resources
    cleanup_memory_pools();
    destroy_cuda_streams();
    
    if (processing_times) {
        delete[] processing_times;
        processing_times = nullptr;
    }
    
    // Reset CUDA device
    hipDeviceReset();
    
    initialized = false;
    std::cout << "🛑 CUDA frame processor shutdown complete" << std::endl;
}

bool CudaFrameProcessor::process_frame_async(const unsigned char* input, int width, int height,
                                           unsigned char* output, int output_width, int output_height,
                                           int stream_id) {
    if (!initialized || stream_id >= config.max_concurrent_streams) {
        return false;
    }
    
    // Safety checks
    if (!input || !output || width <= 0 || height <= 0 || 
        output_width <= 0 || output_height <= 0) {
        return false;
    }
    
    // Dimension safety checks (prevent OpenCV assertion failures)
    if (width > 32767 || height > 32767 || output_width > 32767 || output_height > 32767) {
        std::cerr << "Frame dimensions exceed safe limits" << std::endl;
        return false;
    }
    
    hipStream_t stream = (hipStream_t)cuda_streams[stream_id];
    
    // Get appropriate frame buffer
    CudaFrameBuffer* buffer = allocate_frame_buffer(width, height, 3);
    if (!buffer) {
        return false;
    }
    
    // Calculate memory requirements
    size_t input_size = width * height * 3;
    size_t output_size = output_width * output_height * 3;
    
    // Copy input to GPU (async)
    CUDA_CHECK(hipMemcpyAsync(buffer->d_input, input, input_size, 
                              hipMemcpyHostToDevice, stream));
    
    // Configure kernel launch parameters
    dim3 block_size(16, 16);
    dim3 grid_size((output_width + block_size.x - 1) / block_size.x,
                   (output_height + block_size.y - 1) / block_size.y);
    
    // Launch resize kernel
    cuda_resize_bilinear_kernel<<<grid_size, block_size, 0, stream>>>(
        (const unsigned char*)buffer->d_input,
        (unsigned char*)buffer->d_output,
        width, height, output_width, output_height, 3,
        buffer->pitch, buffer->pitch
    );
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());
    
    // Copy result back to host (async)
    CUDA_CHECK(hipMemcpyAsync(output, buffer->d_output, output_size,
                              hipMemcpyDeviceToHost, stream));
    
    return true;
}

bool CudaFrameProcessor::wait_for_completion(int stream_id) {
    if (!initialized || stream_id >= config.max_concurrent_streams) {
        return false;
    }
    
    hipStream_t stream = (hipStream_t)cuda_streams[stream_id];
    CUDA_CHECK(hipStreamSynchronize(stream));
    return true;
}

bool CudaFrameProcessor::process_frame_batch(unsigned char** inputs, int batch_size,
                                           int width, int height, unsigned char** outputs,
                                           int output_width, int output_height) {
    if (!initialized || batch_size <= 0) {
        return false;
    }
    
    // Process each frame in the batch using different streams
    for (int i = 0; i < batch_size; i++) {
        int stream_id = i % config.max_concurrent_streams;
        
        if (!process_frame_async(inputs[i], width, height, outputs[i], 
                               output_width, output_height, stream_id)) {
            return false;
        }
    }
    
    // Wait for all streams to complete
    for (int i = 0; i < config.max_concurrent_streams; i++) {
        wait_for_completion(i);
    }
    
    return true;
}

CudaFrameBuffer* CudaFrameProcessor::allocate_frame_buffer(int width, int height, int channels) {
    // Find available buffer or create new one
    for (auto& buffer : frame_buffers) {
        if (buffer.width == width && buffer.height == height && buffer.channels == channels) {
            return &buffer;
        }
    }
    
    // Create new buffer
    CudaFrameBuffer buffer;
    buffer.width = width;
    buffer.height = height;
    buffer.channels = channels;
    
    // Calculate pitched memory requirements
    size_t width_bytes = width * channels;
    CUDA_CHECK(hipMallocPitch(&buffer.d_input, &buffer.pitch, width_bytes, height));
    CUDA_CHECK(hipMallocPitch(&buffer.d_output, &buffer.pitch, width_bytes, height));
    CUDA_CHECK(hipMallocPitch(&buffer.d_temp, &buffer.pitch, width_bytes, height));
    
    frame_buffers.push_back(buffer);
    return &frame_buffers.back();
}

void CudaFrameProcessor::release_frame_buffer(CudaFrameBuffer* buffer) {
    if (!buffer) return;
    
    // In a real implementation, this would return buffer to pool
    // For now, buffers are cleaned up in shutdown()
}

float CudaFrameProcessor::get_average_processing_time() const {
    if (!processing_times) return 0.0f;
    
    float sum = 0.0f;
    int count = 0;
    
    for (int i = 0; i < 100; i++) {
        if (processing_times[i] > 0.0f) {
            sum += processing_times[i];
            count++;
        }
    }
    
    return count > 0 ? sum / count : 0.0f;
}

float CudaFrameProcessor::get_gpu_utilization() const {
    // This would use NVML to get actual GPU utilization
    // For now, return estimated utilization based on processing times
    float avg_time = get_average_processing_time();
    float target_time = 1000.0f / 30.0f; // 30 FPS target
    
    return std::min(100.0f, (avg_time / target_time) * 100.0f);
}

size_t CudaFrameProcessor::get_memory_usage() const {
    size_t total_usage = 0;
    
    for (const auto& buffer : frame_buffers) {
        size_t buffer_size = buffer.pitch * buffer.height * 3; // 3 buffers per frame buffer
        total_usage += buffer_size;
    }
    
    return total_usage;
}

bool CudaFrameProcessor::allocate_memory_pools() {
    // Allocate main memory pool
    memory_pool_size = config.memory_pool_size_mb * 1024 * 1024;
    
    CUDA_CHECK(hipMalloc(&memory_pool, memory_pool_size));
    
    std::cout << "Allocated CUDA memory pool: " << config.memory_pool_size_mb << " MB" << std::endl;
    return true;
}

void CudaFrameProcessor::cleanup_memory_pools() {
    // Free all frame buffers
    for (auto& buffer : frame_buffers) {
        if (buffer.d_input) hipFree(buffer.d_input);
        if (buffer.d_output) hipFree(buffer.d_output);
        if (buffer.d_temp) hipFree(buffer.d_temp);
    }
    frame_buffers.clear();
    
    // Free main memory pool
    if (memory_pool) {
        hipFree(memory_pool);
        memory_pool = nullptr;
    }
}

bool CudaFrameProcessor::create_cuda_streams() {
    for (int i = 0; i < config.max_concurrent_streams && i < 8; i++) {
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        cuda_streams[i] = (void*)stream;
    }
    
    std::cout << "Created " << config.max_concurrent_streams << " CUDA streams" << std::endl;
    return true;
}

void CudaFrameProcessor::destroy_cuda_streams() {
    for (int i = 0; i < 8; i++) {
        if (cuda_streams[i]) {
            hipStreamDestroy((hipStream_t)cuda_streams[i]);
            cuda_streams[i] = nullptr;
        }
    }
}

void CudaFrameProcessor::cleanup_cuda_streams() {
    // Alias for destroy_cuda_streams for compatibility
    destroy_cuda_streams();
}

// C API Implementation
extern "C" {
    DLLEXPORT CudaFrameProcessor* create_cuda_processor(CudaConfig config) {
        CudaFrameProcessor* processor = new CudaFrameProcessor(config);
        if (!processor->initialize()) {
            delete processor;
            return nullptr;
        }
        return processor;
    }
    
    DLLEXPORT void destroy_cuda_processor(CudaFrameProcessor* processor) {
        if (processor) {
            delete processor;
        }
    }
    
    DLLEXPORT int cuda_process_frame(CudaFrameProcessor* processor,
                                   unsigned char* input, int width, int height,
                                   unsigned char* output, int output_width, int output_height) {
        if (!processor) return 0;
        
        if (processor->process_frame_async(input, width, height, output, output_width, output_height, 0)) {
            return processor->wait_for_completion(0) ? 1 : 0;
        }
        return 0;
    }
    
    DLLEXPORT int cuda_process_frame_batch(CudaFrameProcessor* processor,
                                         unsigned char** inputs, int batch_size,
                                         int width, int height,
                                         unsigned char** outputs,
                                         int output_width, int output_height) {
        if (!processor) return 0;
        
        return processor->process_frame_batch(inputs, batch_size, width, height, 
                                            outputs, output_width, output_height) ? 1 : 0;
    }
    
    DLLEXPORT float cuda_get_processing_time(CudaFrameProcessor* processor) {
        return processor ? processor->get_average_processing_time() : 0.0f;
    }
    
    DLLEXPORT float cuda_get_gpu_utilization(CudaFrameProcessor* processor) {
        return processor ? processor->get_gpu_utilization() : 0.0f;
    }
    
    DLLEXPORT int cuda_get_memory_usage_mb(CudaFrameProcessor* processor) {
        if (!processor) return 0;
        return (int)(processor->get_memory_usage() / (1024 * 1024));
    }
    
    DLLEXPORT int cuda_get_device_count() {
        int count = 0;
        hipGetDeviceCount(&count);
        return count;
    }
    
    DLLEXPORT int cuda_get_device_memory_mb(int device_id) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, device_id) == hipSuccess) {
            return (int)(prop.totalGlobalMem / (1024 * 1024));
        }
        return 0;
    }
    
    DLLEXPORT int cuda_set_device(int device_id) {
        return hipSetDevice(device_id) == hipSuccess ? 1 : 0;
    }
    
    DLLEXPORT int cuda_is_available() {
        int count = 0;
        return hipGetDeviceCount(&count) == hipSuccess && count > 0 ? 1 : 0;
    }
}
